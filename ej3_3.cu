#include "hip/hip_runtime.h"
#include "./common.h"
#include "./generator.cuh"

// calculate_sum_of_tan_xy: Calculates sum of tan for every row in a cube with number of points num_points
__global__ void calculate_sum_of_tan_yz(unsigned int num_points, Point3D *points, double *sum_result) {
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int k = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ Point3D *point;
  double sum = 0;
  for (unsigned int i = 0; i < num_points; i++) {
    point = &points[k + j * num_points + i * num_points * num_points];
    sum += tan(point->x + point->y + point->z);
  }

  sum_result[j + k * num_points] = sum;
}

int main() {
  // Builds cube of points with space of BIG_POINT_SIZE
  unsigned int num_points_3d = trunc(SQUARE_LENGTH / BIG_POINT_SIZE);

  Point3D *d_points_3d;
  CUDA_CHK(hipMalloc(&d_points_3d, num_points_3d * num_points_3d * num_points_3d * sizeof(Point3D)));

  // Generates points inside a cube
  dim3 block_dim_cube(1, 1, 1);
  dim3 grid_dim_cube(num_points_3d, num_points_3d, num_points_3d);
  generate_cube<<<grid_dim_cube, block_dim_cube>>>(d_points_3d, num_points_3d);
  CUDA_CHK(hipGetLastError());
  CUDA_CHK(hipDeviceSynchronize());

  // Point3D *points_3d = (Point3D *)malloc(num_points_3d * num_points_3d * num_points_3d * sizeof(Point3D));
  // CUDA_CHK(hipMemcpy(points_3d, d_points_3d, num_points_3d * num_points_3d * num_points_3d * sizeof(Point3D), hipMemcpyDeviceToHost));

  // Execute tan calculation
  double *gpu_result;
  CUDA_CHK(hipMalloc((void **)&gpu_result, num_points_3d * sizeof(double)));

  calculate_sum_of_tan_yz<<<grid_dim_cube, block_dim_cube>>>(num_points_3d, d_points_3d, gpu_result);
  CUDA_CHK(hipGetLastError());
  CUDA_CHK(hipDeviceSynchronize());

  double *result = (double *)malloc(num_points_3d * sizeof(double));
  CUDA_CHK(hipMemcpy(result, gpu_result, num_points_3d * sizeof(double), hipMemcpyDeviceToHost));

  print_matrix_of_points(result, 8);

  free(result);
  CUDA_CHK(hipFree(gpu_result));
}
